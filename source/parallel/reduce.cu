#include "hip/hip_runtime.h"
#include "fmt/color.h"
#include "lolly/parallel/basic.h"

#include <cmath>
#include <hip/hip_runtime.h>

// please refer to https://zhuanlan.zhihu.com/p/654027980 for more details

using namespace Lolly::parallel;

template <typename Op, typename Type>
__device__ Type operator()(Type a, Type b, Op op) {
  return op(a, b);
}

template <typename Type> struct Sum {
  __device__ Type operator()(Type a, Type b) { return a + b; }
};

template <typename Type> struct Max {
  __device__ Type operator()(Type a, Type b) { return max(a, b); }
};

template <typename Type> struct Min {
  __device__ Type operator()(Type a, Type b) { return min(a, b); }
};

template <typename Op>
__global__ void reduce(float *input, float *output, Op op) {
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // atomicAdd(output, input[idx] + input[idx + offset]);
  int tid = threadIdx.x;
  float *dv = input + blockDim.x * blockIdx.x;
  for (int offset = blockDim >> 1; 0 < offset; offset >> 1) {
    if (tid < offset) {
      op(dv[tid], dv[tid + offset]);
    }
    __syncthreads();
  }
  if (0 == tid) {
    //*output+=dv[0];//may have problem when different thread blocks
    atomicAdd(output, dv[0]);
  }
}

void Lolly::reduce(float *input, float **out, int size, ReduceType::Type type) {
  if (nullptr == input || out == nullptr || size <= 0) {
    return;
  }
  if (nullptr == *out) {
    *out = new float[size];
    fmt::print(fg(fmt::color::yellow) | fmt::emphasis::bold,
               "out is not allocated outside, allocate it inside reduce");
  }

  hipError_t err = hipSuccess;

  // Allocate device memory
  float *d_input = nullptr;
  float *d_output = nullptr;
  err = hipMalloc((void **)&d_input, sizeof(float));
  if (err != hipSuccess) {
    fprintf(
        stderr,
        "Failed to allocate device memory for input array (error code %s)!\n",
        hipGetErrorString(err));
    return;
  }

  err = hipMalloc((void **)&d_output, sizeof(float));
  if (err != hipSuccess) {
    fprintf(
        stderr,
        "Failed to allocate device memory for output array (error code %s)!\n",
        hipGetErrorString(err));
    hipFree(d_input);
    return;
  }
  hipMemset(d_output, 0, 1);

  hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

  int MAX_BLOCK_THREADS = 256;

  // int half_len = std::ceil(size * 1.0 / 2); // TODO:变为2的倍数
  int NUM_OF_BLOCKS = (size + MAX_BLOCK_THREADS - 1) / MAX_BLOCK_THREADS;

  dim3 blockDim(MAX_BLOCK_THREADS, 1, 1);
  dim3 gridDim(NUm_OF_BLOCKS, 1, 1);
  // Launch the kernel
  switch (type) {
  case ReduceType::SUM: {
    reduce<<<gridDim, blockDim>>>(d_input, d_output, Sum());
    break;
  }
  case ReduceType::MAX: {
    reduce<<<gridDim, blockDim>>>(d_input, d_output, Max());
    break;
  }
  case ReduceType::MIN: {
    reduce<<<gridDim, blockDim>>>(d_input, d_output, Min());
    break;
  }
  default:
    fprintf(stderr, "Invalid reduce type!\n");
    return;
  }
  hipFree(&d_input);
  hipFree(d_output);
}